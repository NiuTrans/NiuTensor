#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * $Created by: Xu Chen (email: hello_master1954@163.com) 2018-09-17
 */

#ifndef __CROSSENTROPY_CUH__
#define __CROSSENTROPY_CUH__

#include "../XTensor.h"
#include "../XDevice.h"
#include "CrossEntropy.cuh"
#include "CrossEntropy.h"
#include "../core/arithmetic/Div.h"
#include "../core/arithmetic/Multiply.h"
#include "../core/arithmetic/MultiplyDim.h"
#include "../core/math/Unary.h"
#include "../core/math/ScaleAndShift.h"
#include "../core/reduce/ReduceSum.h"
#include "../core/reduce/ReduceSumAll.h"
#include "../core/shape/Transpose.h"
#include "../core/shape/Unsqueeze.h"
#include "../core/shape/IsSameShaped.h"

namespace nts{ // namespace nts(NiuTrans.Tensor)

/*
compute the cross entropy loss (cuda version) 
loss = sum_{i} (-gold_i * log(output_i))
where gold and output are distributions 
        
>> output - model prediction
>> gold - gold standard
>> loss - compute loss
>> weight - a rescaling weight given to each class
>> padding - specify a target value that is ignored and does not contribute to the loss computation
>> leadingDim - the leading dimension for the output
*/
void _CudaCrossEntropyFast(const XTensor * output, const XTensor * gold,
                           XTensor * loss, const XTensor * weight, 
                           const XTensor * padding, int leadingDim)
{
    int n = leadingDim < 0 ? output->order - 1 : leadingDim;
    
    if (output->mem != NULL) {
        output->mem->LockBuf();
    }
    XTensor * interBuf1 = NewTensorBufV2(output, output->devID, output->mem);
    XTensor * interBuf2 = NewTensorBufV2(output, output->devID, output->mem);
    
    _Log(output, interBuf1);
    _Multiply(gold, interBuf1, interBuf2);

    if(weight != NULL)
        _MultiplyDimMe(interBuf2, weight, n);
    _NegateMe(interBuf2);
    _ReduceSum(interBuf2, loss, n);
    
    if(padding != NULL)
        _MultiplyMe(loss, padding);

    DelTensorBuf(interBuf2);
    DelTensorBuf(interBuf1);
    if (output->mem != NULL) {
        output->mem->UnlockBuf();
    }
}

/*
compute the cross entropy loss (scalar version) 

loss = sum_{i} (-gold_i * log(output_i))
where gold and output are distributions 
        
>> output - model prediction
>> gold - gold standard
>> reduceWay - loss compute way, sum or mean
>> weight - a rescaling weight given to each class
>> padding - specify a target value that is ignored and does not contribute to the loss computation
>> leadingDim - the leading dimension for the output
<< return - the cross entropy loss that is a scalar
*/
DTYPE _CudaCrossEntropyFast(const XTensor * output, const XTensor * gold,
                            LOSS_COMPUTE_WAY reduceWay, const XTensor * weight,
                            const XTensor * padding, int leadingDim)
{
    DTYPE loss = 0;

    int order = output->order;
    int n = leadingDim < 0 ? output->order - 1 : leadingDim;
    int leadingDimSize = output->GetDim(n);

    CheckNTErrors(n >= 0 && n < output->order, 
                 "Wrong leadingDim!");
    CheckNTErrors(_IsSameShaped(output, gold), 
                 "The output tensor and gold tensor must be of the same size!");
    CheckNTErrors(weight == NULL || weight->unitNum == leadingDimSize, 
                 "Wrong weight tensor!");
    CheckNTErrors(padding == NULL || padding->order == output->order - 1, 
                 "Wrong padding tensor!");
    CheckNTErrors(gold->dataType == DEFAULT_DTYPE && output->dataType == DEFAULT_DTYPE, 
                 "TODO!");
    
    int * dimSize = new int[output->order - 1];
    for (int i = 0; i < order; i++) {
        if(i < n)
            dimSize[i] = output->dimSize[i];
        else if(i > n)
            dimSize[i - 1] = output->dimSize[i];
    }

    if (output->mem != NULL) {
        output->mem->LockBuf();
    }
    XTensor * lossBuf = NewTensorBufV2(output->order - 1, dimSize, output->dataType, output->denseRatio, 
                                     output->devID, output->mem);

    _CudaCrossEntropyFast(output, gold, lossBuf, weight, padding, leadingDim);

    _ReduceSumAll(lossBuf, &loss);

    if(reduceWay == REDUCE_MEAN) {
        DTYPE nonZeroNum;
        if(padding == NULL) {
            nonZeroNum = (DTYPE)lossBuf->unitNum;
        }
        else {
            if ((padding->mem != NULL) && (padding->mem != output->mem)) {
                padding->mem->LockBuf();
            }
            XTensor * tmp = NewTensorBufV2(padding, padding->devID, padding->mem);
            _IsNonZero(padding, tmp);
            _ReduceSumAll(tmp, &nonZeroNum);
            DelTensorBuf(tmp);
            if ((padding->mem != NULL) && (padding->mem != output->mem)) {
                padding->mem->UnlockBuf();
            }
        }

        loss = loss / nonZeroNum;
    }
    else if(reduceWay == REDUCE_SUM) {
        /* don't need to do anything */
    }
    else {
        ShowNTErrors("TODO");
    }

    delete[] dimSize;
    DelTensorBuf(lossBuf);
    if (output->mem != NULL) {
        output->mem->UnlockBuf();
    }

    return loss;
}

/* 
backward computation of cross entropy function 

loss = sum_{i} (-t_i * log(y_i))
dE/dy_i = -t_i / y_i
where E is the error(loss) function that measure the errors in y
with respect to gold standard, and y this the model output

>> dedy - dE/dy (for return)
>> output - model prediction
>> gold - gold standard
>> weight - a rescaling weight given to each class
>> padding - specify a target value that is ignored and does not contribute to the loss computation
>> leadingDim - the leading dimension for the output
*/
void _CudaCrossEntropyBackward(XTensor * dedy, const XTensor * output, 
                               const XTensor * gold, const XTensor * weight,
                               XTensor * padding, int leadingDim)
{
    int n = leadingDim < 0 ? output->order - 1 : leadingDim;
    
    _Div(gold, output, dedy);
    _NegateMe(dedy);
    if(weight != NULL)
        _MultiplyDimMe(dedy, weight, n);
    if(padding != NULL) {
        int paddingOrder = padding->order;
        int * paddingDims = new int[paddingOrder];
        memcpy(paddingDims, padding->dimSize, padding->order * sizeof(int));
        padding->Reshape(padding->unitNum);

        int order = dedy->order;
        int * dims = new int[order];
        memcpy(dims, dedy->dimSize, dedy->order * sizeof(int));
        dedy->Reshape(dedy->unitNum/dedy->GetDim(n), dedy->GetDim(n));
        _MultiplyDimMe(dedy, padding, 0);

        padding->Reshape(paddingOrder, paddingDims);
        dedy->Reshape(order, dims);

        delete[] paddingDims;
        delete[] dims;
    }

    if(padding != NULL) {
        XTensor * tmp = NewTensor(padding);
        _IsNonZero(padding, tmp);
        DTYPE nonZeroNum;
        _ReduceSumAll(tmp, &nonZeroNum);
        _ScaleAndShiftMe(dedy, (DTYPE)1.0/nonZeroNum);
        delete tmp;
    }
    else {
        int num = dedy->unitNum / dedy->GetDim(n);
        _ScaleAndShiftMe(dedy, (DTYPE)1.0/(DTYPE)num);
    }

}

} // namespace nts(NiuTrans.Tensor)

#endif // __CROSSENTROPY_CUH__