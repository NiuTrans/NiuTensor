#include "hip/hip_runtime.h"
/* NiuTrans.Tensor - an open-source tensor library
 * Copyright (C) 2017, Natural Language Processing Lab, Northeastern University. 
 * All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
* $Created by: XIAO Tong (email: xiaotong@mail.neu.edu.cn) 2018-04-24
*/

#include "../../XDevice.h"
#include "../../XTensor.h"
#include "../../XUtility.h"
#include "ReduceMax.h"
#include "ReduceMax.cuh"

namespace nts{ // namespace nts(NiuTrans.Tensor)

#ifdef USE_CUDA


/*
use PTX code to reduce float data
*/
#define SHLFUNCFLOAT(funcName, reducePTXOp)                    \
__device__ __forceinline__                                     \
float funcName(float input)                                    \
{                                                              \
    float output;                                              \
    asm volatile(                                              \
        "{"                                                    \
        ".reg .f32 r0;"                                        \
        ".reg .pred p;"                                        \
        "shfl.sync.down.b32  r0, %1, 0x10, 0x1f,0xffffffff;"   \
        "setp."#reducePTXOp".f32    p,%1,r0;"                  \
        "@p mov.f32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x8, 0xf,0xffffffff;"     \
        "setp."#reducePTXOp".f32    p,%1,r0;"                  \
        "@p mov.f32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x4, 0x7,0xffffffff;"     \
        "setp."#reducePTXOp".f32    p,%1,r0;"                  \
        "@p mov.f32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x2, 0x3,0xffffffff;"     \
        "setp."#reducePTXOp".f32    p,%1,r0;"                  \
        "@p mov.f32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x1, 0x1,0xffffffff;"     \
        "setp."#reducePTXOp".f32    p, %1, r0; "               \
        "@p mov.f32     %1,r0;"                                \
        "mov.f32        %0,%1;"                                \
        "}"                                                    \
        : "=f"(output) : "f"(input));                          \
    return output;                                             \
}

SHLFUNCFLOAT(shflDownReduceMax, lt)
SHLFUNCFLOAT(shflDownReduceMin, gt)

/*
use PTX code to reduce int data
*/
#define SHLFUNCINT(funcName, reducePTXOp)                      \
__device__ __forceinline__                                     \
int funcName(int input)                                        \
{                                                              \
    int output;                                                \
    asm volatile(                                              \
        "{"                                                    \
        ".reg .s32 r0;"                                        \
        ".reg .pred p;"                                        \
        "shfl.sync.down.b32  r0, %1, 0x10, 0x1f,0xffffffff;"   \
        "setp."#reducePTXOp".s32    p,%1,r0;"                  \
        "@p mov.s32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x8, 0xf,0xffffffff;"     \
        "setp."#reducePTXOp".s32    p,%1,r0;"                  \
        "@p mov.s32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x4, 0x7,0xffffffff;"     \
        "setp."#reducePTXOp".s32    p,%1,r0;"                  \
        "@p mov.s32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x2, 0x3,0xffffffff;"     \
        "setp."#reducePTXOp".s32    p,%1,r0;"                  \
        "@p mov.s32     %1,r0;"                                \
        "shfl.sync.down.b32  r0, %1, 0x1, 0x1,0xffffffff;"     \
        "setp."#reducePTXOp".s32    p, %1, r0; "               \
        "@p mov.s32     %1,r0;"                                \
        "mov.s32        %0,%1;"                                \
        "}"                                                    \
        : "=r"(output) : "r"(input));                          \
    return output;                                             \
}

SHLFUNCINT(shflDownReduceMax, lt)
SHLFUNCINT(shflDownReduceMin, gt)

/* 
reduce a tensor to another that keeps the max value along a dimension  - slow version
Given a block of data, we go over each dimension i in the stride and we have
sum_i = max_{0<=j<strideNum} input_{i,j}
where we can view the block as a matrix and input_{i,j} represent the item at the
crossing of the i-th columne and the j-th row.
>> input - the input array (representing a tensor)
>> output - the sum over each block. NOTE: output is also an array
>> stride - stride that we need to move to the next item
>> strideNum - how many strides we need to finish the reduce
>> reducedStrideNum - the number of strides after reducation 
>> blockSize - size of the block (i.e., stride * strideNum)
>> blockNum - how many blocks
*/
#define KERNELREDUCEFUN3(funName, opName, initData)                                                         \
 __global__                                                                                                 \
void funName(DTYPE * input, DTYPE * output,                                                                 \
                     int stride, int strideNum, int reducedStrideNum,                                       \
                     int blockSize, int blockNum)                                                           \
{                                                                                                           \
    __shared__ DTYPE iData[MAX_CUDA_THREAD_NUM_PER_BLOCK * MIN_CUDA_SHARED_MEM_COL_SIZE/2];                 \
                                                                                                            \
    int idx = threadIdx.x * blockDim.y + threadIdx.y;                                                       \
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;                                                   \
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;                                                   \
                                                                                                            \
    if(i >= stride * blockNum)                                                                              \
        return;                                                                                             \
                                                                                                            \
    __syncthreads();                                                                                        \
                                                                                                            \
    int k = i / stride;                                                                                     \
    int iOffset = i % stride;                                                                               \
                                                                                                            \
    DTYPE value = (i < stride * blockNum && j < strideNum) ?                                                \
                   input[blockSize * k + stride * j + iOffset] : initData;                                  \
                                                                                                            \
    /* load data into the shared mem */                                                                     \
    iData[threadIdx.x * blockDim.y + threadIdx.y] = value;                                                  \
                                                                                                            \
    __syncthreads();                                                                                        \
                                                                                                            \
    /* do reduction in shared mem */                                                                        \
    for (unsigned int s = blockDim.y/2; s > 0; s >>= 1){                                                    \
        if(threadIdx.y < s){                                                                                \
            iData[idx] = opName(iData[idx + s], iData[idx]);                                                \
        }                                                                                                   \
                                                                                                            \
        __syncthreads();                                                                                    \
    }                                                                                                       \
                                                                                                            \
    /* write result for this block to the output array */                                                   \
    if (threadIdx.y == 0 && blockIdx.y < reducedStrideNum)                                                  \
        output[(k * reducedStrideNum + blockIdx.y) * stride + iOffset] = iData[threadIdx.x * blockDim.y];   \
                                                                                                            \
}

KERNELREDUCEFUN3(KernelReduceMax, MAX, FLOAT_MIN)
KERNELREDUCEFUN3(KernelReduceMin, MIN, MAX_FLOAT)

/*
reduce a tensor to another that keeps the max value along a dimension  - slow version
Given a block of data, we go over each dimension i in the stride and we have
sum_i = max_{0<=j<strideNum} input_{i,j}
where we can view the block as a matrix and input_{i,j} represent the item at the
crossing of the i-th columne and the j-th row.
>> input - the input array (representing a tensor)
>> output - the sum over each block. NOTE: output is also an array
>> stride - stride that we need to move to the next item
>> strideNum - how many strides we need to finish the reduce
>> reducedStrideNum - the number of strides after reducation
>> blockSize - size of the block (i.e., stride * strideNum)
>> blockNum - how many blocks
*/
__global__
void KernelReduceMax(__half * input, __half * output,
                     int stride, int strideNum, int reducedStrideNum,
                     int blockSize, int blockNum)
{
    int idx = threadIdx.x * blockDim.y + threadIdx.y;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i >= stride * blockNum)
        return;

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    __shared__ __half iData[MAX_CUDA_THREAD_NUM_PER_BLOCK * MIN_CUDA_SHARED_MEM_COL_SIZE / 2];
#else
    __shared__ DTYPE iData[MAX_CUDA_THREAD_NUM_PER_BLOCK * MIN_CUDA_SHARED_MEM_COL_SIZE / 2];
#endif

    __syncthreads();

    int k = i / stride;
    int iOffset = i % stride;

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    __half value = (i < stride * blockNum && j < strideNum) ?
         input[blockSize * k + stride * j + iOffset] : __half(FLOAT16_MIN);
#else
    DTYPE value = (i < stride * blockNum && j < strideNum) ?
        __half2float(input[blockSize * k + stride * j + iOffset]) : FLOAT_MIN;
#endif

    /* load data into the shared mem */
    iData[threadIdx.x * blockDim.y + threadIdx.y] = value;

    __syncthreads();

    /* do reduction in shared mem */
    for (unsigned int s = blockDim.y / 2; s > 0; s >>= 1) {
        if (threadIdx.y < s && iData[idx] < iData[idx + s]) {
            iData[idx] = iData[idx + s];
        }

        __syncthreads();
    }

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    /* write result for this block to the output array */
    if (threadIdx.y == 0 && blockIdx.y < reducedStrideNum)
        output[(k * reducedStrideNum + blockIdx.y) * stride + iOffset] = iData[threadIdx.x * blockDim.y];
#else
    /* write result for this block to the output array */
    if (threadIdx.y == 0 && blockIdx.y < reducedStrideNum)
        output[(k * reducedStrideNum + blockIdx.y) * stride + iOffset] = __half(iData[threadIdx.x * blockDim.y]);
#endif

 }

/* 
reduce a tensor to another that keeps the max value along a dimension  - fast version
>> input - the input array (representing a tensor)
>> output - the sum over each block. NOTE: output is also an array
>> stride - stride that we need to move to the next item
>> strideNum - how many strides we need to finish the reduce
>> reducedStrideNum - the number of strides after reducation 
>> blockSize - size of the block (i.e., stride * strideNum)
>> blockNum - how many blocks
*/
#define KERNELREDUCEFUN4(funName, opName, opFuncName, initData)                                            \
template <unsigned int goodSize> __global__                                                                \
void funName(DTYPE * input, DTYPE * output,                                                    \
                         int stride, int strideNum, int reducedStrideNum,                                  \
                         int blockSize, int blockNum)                                                      \
{                                                                                                          \
    __shared__ DTYPE iData[MAX_CUDA_THREAD_NUM_PER_BLOCK];                                                 \
                                                                                                           \
    unsigned int tid = threadIdx.y;                                                                        \
    unsigned int j = blockIdx.y * (blockDim.y * 2) + threadIdx.y;                                          \
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;                                                \
                                                                                                           \
    if(i >= stride * blockNum)                                                                             \
        return;                                                                                            \
                                                                                                           \
    __syncthreads();                                                                                       \
                                                                                                           \
    /* first level reduction */                                                                            \
    int k = i / stride;                                                                                    \
    int iOffset = i % stride;                                                                              \
                                                                                                           \
    DTYPE * data = iData + threadIdx.x * blockDim.y;                                                       \
    DTYPE * inputData = input + k * blockSize;                                                             \
    DTYPE value = j < strideNum ? inputData[j * stride + iOffset] : initData;                              \
    DTYPE value2 = j + blockDim.y < strideNum ? inputData[(j + blockDim.y) * stride + iOffset]: initData;  \
                                                                                                           \
    value = opName(value, value2);                                                                         \
    value = opFuncName(value);                                                                             \
    if ((tid & 0x1f) == 0)                                                                                 \
        data[tid / 32] = value;                                                                            \
    __syncthreads();                                                                                       \
                                                                                                           \
    if (tid < 32) {                                                                                        \
        if (tid < blockDim.y / 32)                                                                         \
            value = data[tid];                                                                             \
        else                                                                                               \
            value = initData;                                                                              \
        value = opFuncName(value);                                                                         \
        if (tid == 0 && blockIdx.y < reducedStrideNum)                                                     \
            output[(k * reducedStrideNum + blockIdx.y) * stride + iOffset] = value;                        \
    }                                                                                                      \
}

KERNELREDUCEFUN4(KernelReduceMaxFast, MAX, shflDownReduceMax, FLOAT_MIN)
KERNELREDUCEFUN4(KernelReduceMinFast, MIN, shflDownReduceMin, MAX_FLOAT)

/*
reduce a tensor to another that keeps the max value along a dimension  - fast version
>> input - the input array (representing a tensor)
>> output - the sum over each block. NOTE: output is also an array
>> stride - stride that we need to move to the next item
>> strideNum - how many strides we need to finish the reduce
>> reducedStrideNum - the number of strides after reducation
>> blockSize - size of the block (i.e., stride * strideNum)
>> blockNum - how many blocks
*/
template <unsigned int goodSize> __global__
void KernelReduceMaxFast(__half * input, __half * output,
                         int stride, int strideNum, int reducedStrideNum,
                         int blockSize, int blockNum)
{
    unsigned int tid = threadIdx.y;
    unsigned int j = blockIdx.y * (blockDim.y * 2) + threadIdx.y;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= stride * blockNum)
        return;

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    __shared__ __half iData[MAX_CUDA_THREAD_NUM_PER_BLOCK];
#else
    __shared__ DTYPE iData[MAX_CUDA_THREAD_NUM_PER_BLOCK];
#endif

    __syncthreads();

    /* first level reduction */
    int k = i / stride;
    int iOffset = i % stride;

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    __half * data = iData + threadIdx.x * blockDim.y;
    __half * inputData = input + k * blockSize;
    __half value = j < strideNum ? inputData[j * stride + iOffset] : __half(FLOAT16_MIN);
    __half value2 = j + blockDim.y < strideNum ? inputData[(j + blockDim.y) * stride + iOffset] : __half(FLOAT16_MIN);
#else
    DTYPE * data = iData + threadIdx.x * blockDim.y;
    __half * inputData = input + k * blockSize;
    DTYPE value = j < strideNum ? __half2float(inputData[j * stride + iOffset]) : FLOAT_MIN;
    DTYPE value2 = j + blockDim.y < strideNum ? __half2float(inputData[(j + blockDim.y) * stride + iOffset]) : FLOAT_MIN;
#endif

    /* load data into the shared mem */
    data[tid] = MAX(value, value2);

    __syncthreads();

    /* unroll the warp */

    if (goodSize >= 512) { if (tid < 256) { if (data[tid] < data[tid + 256]) data[tid] = data[tid + 256]; } __syncthreads(); }
    if (goodSize >= 256) { if (tid < 128) { if (data[tid] < data[tid + 128]) data[tid] = data[tid + 128]; } __syncthreads(); }
    if (goodSize >= 128) { if (tid <  64) { if (data[tid] < data[tid +  64]) data[tid] = data[tid +  64]; } __syncthreads(); }
    if (goodSize >=  64) { if (tid <  32) { if (data[tid] < data[tid +  32]) data[tid] = data[tid +  32]; } __syncthreads(); }
    if (goodSize >=  32) { if (tid <  16) { if (data[tid] < data[tid +  16]) data[tid] = data[tid +  16]; } __syncthreads(); }
    if (goodSize >=  16) { if (tid <   8) { if (data[tid] < data[tid +   8]) data[tid] = data[tid +   8]; } __syncthreads(); }
    if (goodSize >=   8) { if (tid <   4) { if (data[tid] < data[tid +   4]) data[tid] = data[tid +   4]; } __syncthreads(); }
    if (goodSize >=   4) { if (tid <   2) { if (data[tid] < data[tid +   2]) data[tid] = data[tid +   2]; } __syncthreads(); }
    if (goodSize >=   2) { if (tid <   1) { if (data[tid] < data[tid +   1]) data[tid] = data[tid +   1]; } __syncthreads(); }

#if __CUDA_ARCH__ >= 530 || !defined(__CUDA_ARCH__)
    /* write result for this block to the output array */
    if (threadIdx.y == 0 && blockIdx.y < reducedStrideNum)
        output[(k * reducedStrideNum + blockIdx.y) * stride + iOffset] = data[0];
#else
    /* write result for this block to the output array */
    if (threadIdx.y == 0 && blockIdx.y < reducedStrideNum)
        output[(k * reducedStrideNum + blockIdx.y) * stride + iOffset] = __float2half(data[0]);
#endif
}

/*
reduce a tensor to another that keeps the max value along a dimension  - simple and fast version
*/
__global__
void KernelReduceMaxSimpleFast(DTYPE * input, DTYPE * output, 
                               int stride, int strideNum, int blockSize, int blockNum)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= stride)
        return;

    int blockIndex = i / blockSize;
    int offset = i % blockSize;

    DTYPE * ip = input + blockIndex * blockSize + offset;
    DTYPE * op = output + blockIndex * stride + offset;

    DTYPE max = DTYPE_MIN;
    if(strideNum % 4 == 0){
        int stride2 = stride + stride;
        int stride3 = stride2 + stride;
        int stride4 = stride3 + stride;
        for(int k = 0; k < blockSize; k += stride4){
            DTYPE m = MAX(MAX(ip[k], ip[k + stride]), MAX(ip[k + stride2], ip[k + stride3]));
            max = MAX(max, m);
        }
    }
    else{
        for (int k = 0; k < blockSize; k += stride)
            max = MAX(max, ip[k]);
    }

    __syncthreads();

    op[offset] = max;
}

/*
according the GPU's sm number allocation warp num
*/
inline void continuousStorageThreadAllocation(dim3& grid, dim3& block, long long vectorNum, int vectorSize)
{
    int warpNum = 4;
    if (vectorNum < 20 * 8){
        warpNum = 8;
        if (vectorNum < 20 * 4){
            warpNum = 16;
            if (warpNum < 20 * 2)
                warpNum = 32;
        }
    }
    int minWarpNum = vectorSize / 32;
    if (vectorSize % 32 != 0) minWarpNum++;
    warpNum = min(warpNum, minWarpNum);

    grid.x = (unsigned int)vectorNum;
    grid.y = 1;
    grid.z = 1;
    block.x = 1;
    block.y = warpNum * 32;
    block.z = 1;
}

/*
adjust threads.x number then we can use warp optimization 
*/
inline void adjustThreadForUseWarpOptimization(dim3& blocks, dim3& threads)
{
    if (threads.x > 1) {
        blocks.x *= threads.x;
        threads.x = 1;
    }
    if (threads.y < 32)
        threads.y = 32;
}

/*
In some case,we use less block to imporve efficiency
*/
#define KERNELREDUCEFUN2(funName, opName, opFuncName, initData)                   \
__global__                                                                        \
void funName(DTYPE * input, DTYPE * output, int strideNum, int blockNum)          \
{                                                                                 \
    int idx = threadIdx.x % 32;                                                   \
    int idy = (blockIdx.x * blockDim.x + threadIdx.x) / 32;                       \
                                                                                  \
    int startIndex = idy * strideNum;                                             \
    DTYPE threadMax = initData;                                                   \
    for (int i = idx; i < strideNum; i += 32) {                                   \
        threadMax = opName(input[startIndex + i], threadMax);                     \
    }                                                                             \
    threadMax = opFuncName(threadMax);                                            \
    if (idx == 0)                                                                 \
        output[idy] = threadMax;                                                  \
}

KERNELREDUCEFUN2(KernelReduceMaxOpLessBlocks, MAX, shflDownReduceMax, FLOAT_MIN)
KERNELREDUCEFUN2(KernelReduceMinOpLessBlocks, MIN, shflDownReduceMin, MAX_FLOAT)


/*
we use PTX code reduce
*/
#define KERNELREDUCEFUN1(funName, opName, opFuncName, initData)                          \
__global__                                                                               \
void funName(DTYPE * input, DTYPE * output,int stride, int strideNum,                    \
                       int reducedStrideNum,int blockSize, int blockNum)                 \
{                                                                                        \
    __shared__ DTYPE iData[MAX_CUDA_THREAD_NUM_PER_BLOCK / 32];                          \
                                                                                         \
    unsigned int tid = threadIdx.y;                                                      \
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;                              \
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;                              \
    if (i >= stride * blockNum)                                                          \
        return;                                                                          \
                                                                                         \
    /* first level reduction */                                                          \
    int k = i / stride;                                                                  \
    int iOffset = i % stride;                                                            \
                                                                                         \
    DTYPE threadMax = initData;                                                          \
                                                                                         \
    DTYPE * data = iData + threadIdx.x * blockDim.y;                                     \
    DTYPE * inputData = input + k * blockSize;                                           \
    for (int it = j; it < strideNum; it += blockDim.y){                                  \
        threadMax = opName(inputData[it * stride + iOffset], threadMax);                 \
    }                                                                                    \
                                                                                         \
    __syncthreads();                                                                     \
    threadMax = opFuncName(threadMax);                                                   \
    if ((tid & 0x1f) == 0)                                                               \
        data[tid / 32] = threadMax;                                                      \
                                                                                         \
    __syncthreads();                                                                     \
    /* use one warp to reduce remaining data */                                          \
    if (tid < 32){                                                                       \
        if (tid < blockDim.y / 32)                                                       \
            threadMax = data[tid];                                                       \
        else threadMax = initData;                                                       \
        threadMax = opFuncName(threadMax);                                               \
        if (tid == 0 && blockIdx.y < reducedStrideNum)                                   \
            output[(k * reducedStrideNum + blockIdx.y) * stride + iOffset] = threadMax;  \
    }                                                                                    \
}

KERNELREDUCEFUN1(KernelReduceMaxOp, MAX, shflDownReduceMax, FLOAT_MIN)
KERNELREDUCEFUN1(KernelReduceMinOp, MIN, shflDownReduceMin, MAX_FLOAT)

/*
get the max-valued items along a dimension of the tensor (cuda version).
For a 1-dimensional data array a,
sum_i = max_{0<=j<strideNum} input_{i,j}
>> input - the input tensor
>> output - the output tensor
>> dim - which dimension to reduce
*/
#define _CUDAREDUCE(_funcName, _reduceFunc1, _reduceFunc2, _reduceFunc3, _reduceFun4)                                                         \
void _funcName(const XTensor * input, XTensor * output, int dim)                                                                              \
{                                                                                                                                             \
    CheckNTErrors(input && output, "Empty input or output tensors!");                                                                         \
    CheckNTErrors(input->order == output->order + 1, "Incorrect tensor sizes!");                                                              \
    CheckNTErrors(input->order > dim && dim >=0, "Illegal dimension to reduce!");                                                             \
    CheckNTErrors(input->dataType == output->dataType, "Unmatched data types!");                                                              \
                                                                                                                                              \
    for(int i = 0; i < input->order; i++){                                                                                                    \
        if(i < dim){                                                                                                                          \
            CheckNTErrors(input->dimSize[i] == output->dimSize[i], "Unmatched tensors!");                                                     \
        }                                                                                                                                     \
        else if(i > dim){                                                                                                                     \
            CheckNTErrors(input->dimSize[i] == output->dimSize[i - 1], "Unmatched tensors!");                                                 \
        }                                                                                                                                     \
    }                                                                                                                                         \
                                                                                                                                              \
    int cudaGridSize[3];                                                                                                                      \
    int cudaBlockSize[3];                                                                                                                     \
    int iter = 0;                                                                                                                             \
    int stride = 1;                                                                                                                           \
    int strideNum = input->dimSize[dim];                                                                                                      \
    int blockSize = 1;                                                                                                                        \
    int blockNum = 1;                                                                                                                         \
                                                                                                                                              \
    for (int i = 0; i < input->order; i++) {                                                                                                  \
        if (i < dim)                                                                                                                          \
            blockNum *= input->dimSize[i];                                                                                                    \
        else if (i > dim)                                                                                                                     \
            stride *= input->dimSize[i];                                                                                                      \
    }                                                                                                                                         \
    blockSize = stride * strideNum;                                                                                                           \
                                                                                                                                              \
    int devID = input->devID;                                                                                                                 \
    int devIDBackup;                                                                                                                          \
    ProtectCudaDev(input->devID, devIDBackup);                                                                                                \
                                                                                                                                              \
    if (stride == 1 && blockNum >= 10) {                                                                                                      \
        dim3 grids;                                                                                                                           \
        dim3 blocks;                                                                                                                          \
        continuousStorageThreadAllocation(grids, blocks, (long long)blockNum, strideNum);                                                     \
        if (blocks.y >= 128) {                                                                                                                \
            _reduceFunc1 <<<grids, blocks >>> ((DTYPE *)input->data, (DTYPE*)output->data, stride, strideNum, grids.y, blockSize, blockNum);  \
        }                                                                                                                                     \
        else {                                                                                                                                \
            if (blockNum % 4 != 0) blockNum = (int)(blockNum / 4) + 1;                                                                        \
            else blockNum = blockNum / 4;                                                                                                     \
            _reduceFunc2 <<<blockNum, 128 >>> ((DTYPE *)input->data, (DTYPE*)output->data, strideNum, blockNum);                              \
        }                                                                                                                                     \
    }                                                                                                                                         \
    else {                                                                                                                                    \
        XMem * mem = input->mem;                                                                                                              \
        GDevs.GetCudaThread2D(devID, strideNum, stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);                                     \
        int bufSize = input->unitSize * cudaGridSize[0] * stride * blockNum * 2;                                                              \
        DTYPE * buf;                                                                                                                          \
        if (mem != NULL) {                                                                                                                    \
            mem->LockBuf();                                                                                                                   \
            buf = (DTYPE*)mem->AllocBuf(mem->devID, bufSize);                                                                                 \
        }                                                                                                                                     \
        else {                                                                                                                                \
            buf = (DTYPE*)XMemAlloc(devID, bufSize);                                                                                          \
        }                                                                                                                                     \
        DTYPE * buf1 = buf;                                                                                                                   \
        DTYPE * buf2 = buf + cudaGridSize[0] * stride * blockNum;                                                                             \
        do {                                                                                                                                  \
            if (input->dataType == DEFAULT_DTYPE) {                                                                                           \
                DTYPE * iData = NULL;                                                                                                         \
                DTYPE * oData = NULL;                                                                                                         \
                if (iter == 0) {                                                                                                              \
                    iData = (DTYPE*)input->data;                                                                                              \
                    oData = buf1;                                                                                                             \
                }                                                                                                                             \
                else if (iter % 2 == 1) {                                                                                                     \
                    iData = buf1;                                                                                                             \
                    oData = buf2;                                                                                                             \
                }                                                                                                                             \
                else {                                                                                                                        \
                    iData = buf2;                                                                                                             \
                    oData = buf1;                                                                                                             \
                }                                                                                                                             \
                                                                                                                                              \
                /* unroll the reduction procedure. The code is messy but it is faster. */                                                     \
                if (strideNum < 32) {                                                                                                         \
                    GDevs.GetCudaThread2D(devID, strideNum, stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);                         \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (DTYPE*)output->data;                                                                                         \
                    _reduceFunc3 <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);                      \
                }                                                                                                                             \
                else if (strideNum < 128) {                                                                                                   \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 64), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);        \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (DTYPE*)output->data;                                                                                         \
                    CheckNTErrors(cudaBlockSize[0] >= 64, "Incorrect thread number when calling the cuda kernel!");                           \
                    adjustThreadForUseWarpOptimization(blocks, threads);                                                                      \
                    _reduceFun4<64> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);                   \
                }                                                                                                                             \
                else if (strideNum < 256) {                                                                                                   \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 128), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);       \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (DTYPE*)output->data;                                                                                         \
                    CheckNTErrors(cudaBlockSize[0] >= 128, "Incorrect thread number when calling the cuda kernel!");                          \
                    adjustThreadForUseWarpOptimization(blocks, threads);                                                                      \
                    _reduceFun4<128> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);                  \
                }                                                                                                                             \
                else if (strideNum < 512) {                                                                                                   \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 256), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);       \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (DTYPE*)output->data;                                                                                         \
                    CheckNTErrors(cudaBlockSize[0] >= 256, "Incorrect thread number when calling the cuda kernel!");                          \
                    adjustThreadForUseWarpOptimization(blocks, threads);                                                                      \
                    _reduceFun4<256> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);                  \
                }                                                                                                                             \
                else {                                                                                                                        \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 512), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);       \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (DTYPE*)output->data;                                                                                         \
                    CheckNTErrors(cudaBlockSize[0] >= 512, "Incorrect thread number when calling the cuda kernel!");                          \
                    adjustThreadForUseWarpOptimization(blocks, threads);                                                                      \
                    _reduceFun4<512> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);                  \
                }                                                                                                                             \
            }                                                                                                                                 \
            else if (input->dataType == X_FLOAT16) {                                                                                          \
                __half * buf1ft16 = (__half *)buf1;                                                                                           \
                __half * buf2ft16 = (__half *)buf2;                                                                                           \
                __half * iData = NULL;                                                                                                        \
                __half * oData = NULL;                                                                                                        \
                if (iter == 0) {                                                                                                              \
                    iData = (__half*)input->data;                                                                                             \
                    oData = buf1ft16;                                                                                                         \
                }                                                                                                                             \
                else if (iter % 2 == 1) {                                                                                                     \
                    iData = buf1ft16;                                                                                                         \
                    oData = buf2ft16;                                                                                                         \
                }                                                                                                                             \
                else {                                                                                                                        \
                    iData = buf2ft16;                                                                                                         \
                    oData = buf1ft16;                                                                                                         \
                }                                                                                                                             \
                                                                                                                                              \
                /* unroll the reduction procedure. The code is messy but it is faster. */                                                     \
                if (strideNum < 32) {                                                                                                         \
                    GDevs.GetCudaThread2D(devID, strideNum, stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);                         \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (__half*)output->data;                                                                                        \
                    KernelReduceMax <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);                   \
                }                                                                                                                             \
                else if (strideNum < 128) {                                                                                                   \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 64), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);        \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (__half*)output->data;                                                                                        \
                    CheckNTErrors(cudaBlockSize[0] >= 64, "Incorrect thread number when calling the cuda kernel!");                           \
                    KernelReduceMaxFast<64> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);           \
                }                                                                                                                             \
                else if (strideNum < 256) {                                                                                                   \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 128), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);       \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (__half*)output->data;                                                                                        \
                    CheckNTErrors(cudaBlockSize[0] >= 128, "Incorrect thread number when calling the cuda kernel!");                          \
                    KernelReduceMaxFast<128> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);          \
                }                                                                                                                             \
                else if (strideNum < 512) {                                                                                                   \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 256), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);       \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (__half*)output->data;                                                                                        \
                    CheckNTErrors(cudaBlockSize[0] >= 256, "Incorrect thread number when calling the cuda kernel!");                          \
                    KernelReduceMaxFast<256> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);          \
                }                                                                                                                             \
                else {                                                                                                                        \
                    GDevs.GetCudaThread2D(devID, MAX(strideNum / 2 + 1, 512), stride * blockNum, MAX_INT, cudaGridSize, cudaBlockSize);       \
                    dim3 blocks(cudaGridSize[1], cudaGridSize[0]), threads(cudaBlockSize[1], cudaBlockSize[0]);                               \
                    if (cudaGridSize[0] == 1)                                                                                                 \
                        oData = (__half*)output->data;                                                                                        \
                    CheckNTErrors(cudaBlockSize[0] >= 512, "Incorrect thread number when calling the cuda kernel!");                          \
                    KernelReduceMaxFast<512> <<<blocks, threads>>> (iData, oData, stride, strideNum, blocks.y, blockSize, blockNum);          \
                }                                                                                                                             \
            }                                                                                                                                 \
                                                                                                                                              \
            strideNum = cudaGridSize[0];                                                                                                      \
            blockSize = cudaGridSize[0];                                                                                                      \
                                                                                                                                              \
            iter++;                                                                                                                           \
                                                                                                                                              \
        } while (strideNum > 1);                                                                                                              \
                                                                                                                                              \
        if (mem != NULL) {                                                                                                                    \
            mem->ReleaseBuf(mem->devID, bufSize);                                                                                             \
            mem->UnlockBuf();                                                                                                                 \
        }                                                                                                                                     \
        else                                                                                                                                  \
            XMemFree(input->devID, buf);                                                                                                      \
    }                                                                                                                                         \
    BacktoCudaDev(input->devID, devIDBackup);                                                                                                 \
}

_CUDAREDUCE(_CudaReduceMax, KernelReduceMaxOp, KernelReduceMaxOpLessBlocks, KernelReduceMax, KernelReduceMaxFast)
_CUDAREDUCE(_CudaReduceMin, KernelReduceMinOp, KernelReduceMinOpLessBlocks, KernelReduceMin, KernelReduceMinFast)


#endif // USE_CUDA

} // namespace nts(NiuTrans.Tensor)